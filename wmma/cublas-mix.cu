
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <iostream>

#include "hipblas.h"

#define CUDA_CHECK(call) \
do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(error) << " at line " << __LINE__ << std::endl; \
        exit(1); \
    } \
} while(0)

#define CUBLAS_CHECK(call) \
do { \
    hipblasStatus_t error = call; \
    if (error != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "CUBLAS error: " << cublasGetStatusString(error) << " at line " << __LINE__ << std::endl; \
        exit(1); \
    } \
} while(0)

void LtHalfMatmul(hipblasLtHandle_t handle,
		int m,
		int n,
		int k,
		const float *alpha,
		// const float *a_scale,
		const __half *A,
		int lda,
		// const float *b_scale,
		const __half *B,
		int ldb,
		// const float *c_scale,
		float *D,
		int ldc,
		// const float *d_scale,
		// float *amax_d,
		void *workspace,
		size_t workspaceSize) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    float beta = 0.0f;

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_16F));
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

//    CUBLAS_CHECK(cublasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale)));
//    CUBLAS_CHECK(cublasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale)));
//    CUBLAS_CHECK(cublasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_C_SCALE_POINTER, &c_scale, sizeof(c_scale)));
//    CUBLAS_CHECK(cublasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale)));
//    CUBLAS_CHECK(cublasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_AMAX_D_POINTER, &amax_d, sizeof(amax_d)));

    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, m, k, lda));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, k, n, ldb));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, m, n, ldc));

    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    CUBLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));
    if (returnedResults == 0) {
        CUBLAS_CHECK(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    CUBLAS_CHECK(hipblasLtMatmul(handle,
		operationDesc,
		alpha,
		A,
		Adesc,
		B,
		Bdesc,
		&beta,
		nullptr,
		Cdesc,
		D,
		Ddesc,
		&heuristicResult.algo,
		workspace,
		workspaceSize,
		0));

    if (Cdesc) CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) CUBLAS_CHECK(hipblasLtMatmulDescDestroy(operationDesc));
}


int main() {
    half *a, *a_d;
    half *b, *b_d;
    float *c, *c_d;

    int m = 4096;
    int n = 4096;
    int k = 4096;

    // Initialize timing infra
    hipEvent_t blas_start, blas_stop;
    CUDA_CHECK(hipEventCreate(&blas_start));
    CUDA_CHECK(hipEventCreate(&blas_stop));
    float blas_eventMs = 1.0f;

    // Allocate host memory
    a = (half*)malloc(m * k * sizeof(half));
    b = (half*)malloc(k * n * sizeof(half));
    c = (float*)malloc(m * n * sizeof(float));

    // Allocate device memory
    CUDA_CHECK(hipMalloc((void**)&a_d, m * k * sizeof(half)));
    CUDA_CHECK(hipMalloc((void**)&b_d, k * n * sizeof(half)));
    CUDA_CHECK(hipMalloc((void**)&c_d, m * n * sizeof(float)));

    for (int i = 0; i < m * k; i++) a[i] = 1.0f;
    for (int i = 0; i < k * n; i++) b[i] = 2.0f;
    for (int i = 0; i < m * n; i++) c[i] = 0.0f;

    CUDA_CHECK(hipMemcpy(a_d, a, m * k * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(b_d, b, k * n * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(c_d, c, m * n * sizeof(half), hipMemcpyHostToDevice));

    hipblasLtHandle_t handle;
    hipblasLtCreate(&handle);

    // Create workspace
    void *workspace;
    CUDA_CHECK(hipMalloc((void**)&workspace, 4194304));

    float alpha = 1.0f;
    CUDA_CHECK(hipEventRecord(blas_start));
    LtHalfMatmul(handle, m, n, k, &alpha, a, m, b, k, c, m, workspace, 4194304);
    CUDA_CHECK(hipEventRecord(blas_stop));
    CUDA_CHECK(hipEventSynchronize(blas_stop));

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(c, c_d, m * n * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventElapsedTime(&blas_eventMs, blas_start, blas_stop));
    std::cout << "cublas: " << blas_eventMs << std::endl;
    for (int i = 0; i < 5; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a);
    free(b);
    free(c);
    hipblasLtDestroy(handle);

    return 0;
}
